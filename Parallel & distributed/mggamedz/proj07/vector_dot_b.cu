/*
	Mkhanyisi Gamedze
	CS336 Prallel & Distributed processing
	project 7
*/
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

/*   tree version  strategy for summing the products */

#define imin(a,b) (a<b?a:b)
#define sum_squares(x)  (x*(x+1)*(2*x+1)/6)

const int N = 33 * 1024;
const int threadsPerBlock = 8192;
const int blocksPerGrid =imin( 32, (N+threadsPerBlock-1) / threadsPerBlock );

// device dot product function
__global__ 
void dot( float *a, float *b, float *c ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float   temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cacheIndex] = temp;

    // synchronize threads in this block
    __syncthreads();

    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}


int main( void ) {
    float   *a, *b, c, *partial_c;
    float   *dev_a, *dev_b, *dev_partial_c;
    float time;
	hipEvent_t start, stop;

    // allocate memory on the CPU side
    a = (float*)malloc( N*sizeof(float) );
    b = (float*)malloc( N*sizeof(float) );
    partial_c = (float*)malloc( blocksPerGrid*sizeof(float) );

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a,N*sizeof(float) ) ;
    hipMalloc( (void**)&dev_b,N*sizeof(float) ) ;
    hipMalloc( (void**)&dev_partial_c,blocksPerGrid*sizeof(float) ) ;

    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N*sizeof(float),hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N*sizeof(float),hipMemcpyHostToDevice );
    
    hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
    
    // compute dot product
    dot<<<blocksPerGrid,threadsPerBlock>>>( dev_a, dev_b,dev_partial_c );
    
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy( partial_c, dev_partial_c,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost );

    // finish up on the CPU side
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }
	
	printf("\nTime naive method:  %.6f ms \n\n", time);
   
    printf( "GPU value %.6g  Expected value %.6g\n", c,2 * sum_squares( (float)(N - 1) ) );

    // free memory on the GPU side
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_partial_c );

    // free memory on the CPU side
    free( a );
    free( b );
    free( partial_c );
}